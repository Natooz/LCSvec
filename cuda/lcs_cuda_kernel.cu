#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAFunctions.h>

namespace {

// TODO: this can probabliy be parallelized using a gpu
template <typename scalar_t>
__device__ int64_t handlePadLen(scalar_t* str, int64_t strLen, int64_t padToken) {
    for (int i=0; i < strLen; i++)
	    if (str[i] == padToken) return i;
    return strLen;
}


template <typename scalar_t>
__global__ void lcs_cuda_kernel(
    scalar_t* const __restrict__ s1,
    scalar_t* const __restrict__ s2,
    int* __restrict__ result,
    int64_t s1Len,
    int64_t s2Len,
    int64_t padToken
) {

    const int batch = blockIdx.x;

    auto s1Batch = s1 + batch * s1Len;
    auto s2Batch = s2 + batch * s2Len;
    auto result_ = result + batch;

    // Handle padding
    s1Len = handlePadLen(s1Batch, s1Len, padToken);
    s2Len = handlePadLen(s2Batch, s2Len, padToken);

    // Zero length
    if (s1Len == 0 || s2Len == 0) {
        *result = torch::empty({0}, {torch::kInt64});
        return;
    }

    // Filling 0's in the matrix
    auto lcsTable = torch::zeros({s1Len, s2Len});

    // Building the matrix in bottom-up way
    for (int i = 1; i <= s1Len; i++) {
        for (int j = 1; j <= s2Len; j++) {
            if (s1[i - 1].equal(s2[j - 1])) {
                lcsTable[i][j] = lcsTable[i - 1][j - 1] + 1;
            } else if ((lcsTable[i - 1][j] >= lcsTable[i][j - 1]).item<bool>()) {
                lcsTable[i][j] = lcsTable[i - 1][j];
            } else {
                lcsTable[i][j] = lcsTable[i][j - 1];
            }
        }
    }

    int index = lcsTable[s1Len][s2Len].item<int>();
    int lcsArr[index + 1];
    int i = s1Len, j = s2Len;
    while (i > 0 && j > 0) {
        if (s1[i - 1].equal(s2[j - 1])) {
            lcsArr[index - 1] = s1[i - 1];
            i--;
            j--;
            index--;
        }
        else if ((lcsTable[i - 1][j] > lcsTable[i][j - 1]).item<bool>())
            i--;
        else
            j--;
    }

    *result = torch::tensor(lcsArr, {torch::kInt64});
    delete(lcsArr);
    delete(lcsTable);
}

}


torch::Tensor lcs_cuda_kernel(
    const torch::Tensor& s1,
    const torch::Tensor& s2,
    torch::Tensor& result,
    int64_t padToken) {

    const auto numBatch = s1.size(0);
    const auto s1Len = s1.size(1);
    const auto s2Len = s2.size(1);

    const int threads = 1;
    const dim3 blocks(numBatch);

    // see https://github.com/pytorch/pytorch/issues/21819
    // to avoid random errors when executing on cuda:1 we need to set the device manually
    c10::cuda::set_device(static_cast<c10::DeviceIndex>(s1.device().index()));

    AT_DISPATCH_ALL_TYPES(
        s1.scalar_type(),
        "lcs_cuda",
        [&] {
             lcs_cuda_kernel<scalar_t><<<numBatch, threads>>>(
                 s1.data<scalar_t>(),
                 s2.data<scalar_t>(),
                 result.data<int>(),
                 s1Len,
                 s2Len,
	             padToken
	         );
        }
    );

    return result;
}
